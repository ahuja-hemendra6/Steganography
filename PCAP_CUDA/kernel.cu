#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>

using namespace std;
using namespace cv;

__global__ void hideIt(int *d_blue, char *d_str, int *d_recv_blue, int tot) {
	int block = blockIdx.x;
	int sstr_len = blockDim.x;
	int id = threadIdx.x;

	int cur = (block * sstr_len) + id;
	char c = d_str[cur];

	if (cur >= tot) {
		return;
	}

	int start = 4 * cur;

	int j = 0;
	d_recv_blue[start + j] = 0;
	for (j = 0; j < 4; j++) {
		d_recv_blue[start + j] = ((d_blue[start + j] >> 2) << 2) | (c & 0x3);
		c = c >> 2;	
	}
	

}

int main() {
	int size = 4, k = 0, i, j, flag;
	int *img_blue;
	int *recv_img_blue;
	int cols, rows, sstr_len;
	char str[1000], recv_str[1000];
	int msg_len, store;
	Mat img;

	int *d_blue, *d_recv_blue;
	char *d_str;

	img = imread("..\\mani.bmp", 1);
	cols = img.cols;
	rows = img.rows;

	img_blue = new int[rows * cols];
	recv_img_blue = new int[cols * rows];

	cout << "Enter the message to be hidden: ";
	cin.getline(str, 1000);
	msg_len = strlen(str);
	sstr_len = ceil((float)msg_len / size);

	k = 0;
	flag = 0;
	store = msg_len;

	cout << store;

	// Burn the size of the message to be written at the end of the image. 
	// This is used to extract how many bytes were written (say n) - from the encoded image.
	// Using this, we can decode the first n/size values from each row
	for (i = rows - 1; i >= 0 && !flag; i--) {
		for (j = cols - 1; j >= 0 && !flag; j--) {
			img.at<Vec3b>(i, j).val[0] = ((img.at<Vec3b>(i, j).val[0] >> 2) << 2) | (store & 0x3);
			store = store >> 2;
			k += 2;
			if (k == 16) {
				flag = 1;
			}
		}
	}

	k = 0;

	// Storing blue values in a single dimension array.
	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < cols; j++)
		{
			Vec3b bgrPixel = img.at<Vec3b>(i, j);
			img_blue[k] = bgrPixel.val[0];
			k++;
		}
	}

	hipMalloc((void **)&d_blue, rows * cols * sizeof(int));
	hipMalloc((void **)&d_str, strlen(str) * sizeof(char));
	hipMalloc((void **)&d_recv_blue, msg_len * 4 * sizeof(int));

	hipMemcpy(d_blue, img_blue, rows * cols * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_str, str, strlen(str) * sizeof(char), hipMemcpyHostToDevice);

	hideIt <<<size, sstr_len>>> (d_blue, d_str, d_recv_blue, msg_len);

	hipMemcpy(recv_img_blue, d_recv_blue, msg_len * 4 * sizeof(int), hipMemcpyDeviceToHost);

	int r = 0;
	int c = 0;
	for (i = 0; i < msg_len * 4; i++, c++) {
		if (c > cols) {
			r++;
			c = 0;
		}
		img.at<Vec3b>(r, c).val[0] = recv_img_blue[i];
	}

	imwrite("..//maniout.bmp", img);
	imshow("Output", img);
	waitKey(0);
	cout << "Done!" << endl;
}